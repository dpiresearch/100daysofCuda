
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *a, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] = i;
}

int main()
{
    int N = 4096;
    int threads = 128;
    int blocks = (N + threads - 1)/threads;
    int *a;

    hipMallocManaged(&a, N * sizeof(int));
    kernel<<<blocks, threads>>>(a, N);
    hipDeviceSynchronize();

    for (int i = 0; i < 20; i++) {
        printf("%d\n", a[i]);
    }

    hipFree(a);
    return 0;

}